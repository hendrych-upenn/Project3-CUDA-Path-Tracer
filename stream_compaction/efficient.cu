#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        using StreamCompaction::Common::kernMapToBoolean;
        using StreamCompaction::Common::kernScatter;

        const int maxBlockSize = 64;

        // assumes padding to block size;
        __global__ void kernBlockScan(int blockLog2Ceil, int* data) {
            int blockOffset = blockDim.x * blockIdx.x;
            int idx = threadIdx.x + 1;
            // upsweep
            for (int i = 1; i <= blockLog2Ceil; i++) {
                int pos = idx * (1 << i) - 1;
                if (pos < blockDim.x) {
                    pos += blockOffset;
                    int offset = 1 << (i - 1);
                    data[pos] = data[pos] + data[pos - offset];
                }
                __syncthreads();
            }
            // downsweep
            // set root to 0

            if (threadIdx.x == blockDim.x - 1) {
                data[blockOffset + blockDim.x - 1] = 0;
            }
            __syncthreads();

            for (int i = blockLog2Ceil; i > 0; i--) {
                int pos = idx * (1 << i) - 1;
                if (pos < blockDim.x) {
                    pos += blockOffset;
                    int offset = 1 << (i - 1);
                    int t = data[pos - offset];
                    data[pos - offset] = data[pos];
                    data[pos] = t + data[pos];
                }

                __syncthreads();
            }
        }

        // assumes padding to block size;
        __global__ void kernBlockScanStoreSum(int blockLog2Ceil, int* data, int* sums) {
            int blockOffset = blockDim.x * blockIdx.x;
            int idx = threadIdx.x + 1;
            // upsweep
            for (int i = 1; i <= blockLog2Ceil; i++) {
                int pos = idx * (1 << i) - 1;
                if (pos < blockDim.x) {
                    pos += blockOffset;
                    int offset = 1 << (i - 1);
                    data[pos] = data[pos] + data[pos - offset];
                }
                __syncthreads();
            }
            // store sum for cross block sum later
            if (threadIdx.x == 0) {
                sums[blockIdx.x] = data[blockOffset + blockDim.x - 1];
            }
            __syncthreads();
            // downsweep
            // set root to 0

            if (threadIdx.x == blockDim.x - 1) {
                data[blockOffset + blockDim.x - 1] = 0;
            }
            __syncthreads();

            for (int i = blockLog2Ceil; i > 0; i--) {
                int pos = idx * (1 << i) - 1;
                if (pos < blockDim.x) {
                    pos += blockOffset;
                    int offset = 1 << (i - 1);
                    int t = data[pos - offset];
                    data[pos - offset] = data[pos];
                    data[pos] = t + data[pos];
                }

                __syncthreads();
            }
        }

        // assumes padding to block size;
        __global__ void kernAddSums(int* data, const int* sums) {
            int idx = blockDim.x * (blockIdx.x) + threadIdx.x;
            data[idx] += sums[blockIdx.x];
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {


            if (n <= 0) {
                return;
            }

            std::vector<int*> scanArrays{};
            std::vector<int> scanArrayLens{};

            // ceil to next maxBlockSize
            int scanArrayLen = divup(n, maxBlockSize) * maxBlockSize;
            while (scanArrayLen > maxBlockSize) {
                int* d_array;
                hipMalloc(&d_array, scanArrayLen * sizeof(int));
                checkCUDAError("hipMalloc scanArrayLen failed");
                hipMemset(d_array, 0, scanArrayLen * sizeof(int));
                scanArrays.push_back(d_array);
                scanArrayLens.push_back(scanArrayLen);
                //fprintf(stderr, "Size %i\n", scanArrayLen);
                // divide by maxBlockSize then ceil to it
                scanArrayLen = divup(scanArrayLen / maxBlockSize, maxBlockSize) * maxBlockSize;
            }
            {
                // scanArrayLen = maxBlockSize now
                int* d_array;
                hipMalloc(&d_array, scanArrayLen * sizeof(int));
                checkCUDAError("hipMalloc scanArrayLen failed");
                hipMemset(d_array, 0, scanArrayLen * sizeof(int));
                scanArrays.push_back(d_array);
                scanArrayLens.push_back(scanArrayLen);
            }

            hipMemcpy(scanArrays[0], idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata failed");

            timer().startGpuTimer();

            for (int i = 0; i < scanArrays.size() - 1; i++) {
                int arrayLen = scanArrayLens[i];
                kernBlockScanStoreSum << < arrayLen / maxBlockSize, maxBlockSize >> > (ilog2ceil(maxBlockSize), scanArrays[i], scanArrays[i + 1]);
                checkCUDAError("kernBlockScanStoreSum failed");
            }
            kernBlockScan << <1, maxBlockSize >> > (ilog2ceil(maxBlockSize), scanArrays.back());
            checkCUDAError("kernBlockScan failed");

            for (int i = scanArrays.size() - 2; i >= 0; i--) {
                int arrayLen = scanArrayLens[i];
                kernAddSums << < arrayLen / maxBlockSize, maxBlockSize >> > (scanArrays[i], scanArrays[i + 1]);
                checkCUDAError("kernAddSums failed");
            }

            timer().endGpuTimer();

            hipMemcpy(odata, scanArrays[0], n * sizeof(int), hipMemcpyDeviceToHost);
            //hipMemcpy(odata, scanArrays.back(), scanArrayLens.back() * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy odata failed");

            for (int i = 0; i < scanArrays.size(); i++) {
                hipFree(scanArrays[i]);
                checkCUDAError("hipFree scanArrays[i] failed");
            }
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            if (n <= 0) {
                return -1;
            }

            int topArrayLen = divup(n, maxBlockSize) * maxBlockSize;

            int* d_idata;
            hipMalloc(&d_idata, topArrayLen * sizeof(int));
            checkCUDAError("hipMalloc d_idata failed");
            hipMemset(d_idata, 0, topArrayLen * sizeof(int));
            checkCUDAError("hipMemset d_idata failed");
            hipMemcpy(d_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy d_idata failed");

            int* d_bools;
            hipMalloc(&d_bools, topArrayLen * sizeof(int));
            checkCUDAError("hipMalloc d_bools failed");


            int* d_odata;
            hipMalloc(&d_odata, n * sizeof(int));
            checkCUDAError("hipMalloc d_odata failed");

            std::vector<int*> scanArrays{};
            std::vector<int> scanArrayLens{};

            // ceil to next maxBlockSize
            int scanArrayLen = topArrayLen;
            while (scanArrayLen > maxBlockSize) {
                int* d_array;
                hipMalloc(&d_array, scanArrayLen * sizeof(int));
                checkCUDAError("kernMapToBoolean failed");
                hipMemset(d_array, 0, scanArrayLen * sizeof(int));
                checkCUDAError("hipMemset d_array failed");
                scanArrays.push_back(d_array);
                scanArrayLens.push_back(scanArrayLen);
                //fprintf(stderr, "Size %i\n", scanArrayLen);
                // divide by maxBlockSize then ceil to it
                scanArrayLen = divup(scanArrayLen / maxBlockSize, maxBlockSize) * maxBlockSize;
            }
            {
                // scanArrayLen = maxBlockSize now
                int* d_array;
                hipMalloc(&d_array, scanArrayLen * sizeof(int));
                checkCUDAError("hipMalloc scanArrayLen failed");
                hipMemset(d_array, 0, scanArrayLen * sizeof(int));
                checkCUDAError("hipMemset d_array failed");
                scanArrays.push_back(d_array);
                scanArrayLens.push_back(scanArrayLen);
            }

            timer().startGpuTimer();

            kernMapToBoolean << <topArrayLen / maxBlockSize, maxBlockSize >> > (topArrayLen, d_bools, d_idata);
            checkCUDAError("kernMapToBoolean failed");

            hipMemcpy(scanArrays[0], d_bools, topArrayLen * sizeof(int), hipMemcpyDeviceToDevice);
            checkCUDAError("hipMemcpy d_bools failed");

            for (int i = 0; i < scanArrays.size() - 1; i++) {
                int arrayLen = scanArrayLens[i];
                kernBlockScanStoreSum << < arrayLen / maxBlockSize, maxBlockSize >> > (ilog2ceil(maxBlockSize), scanArrays[i], scanArrays[i + 1]);
                checkCUDAError("kernBlockScanStoreSum failed");
            }
            kernBlockScan << <1, maxBlockSize >> > (ilog2ceil(maxBlockSize), scanArrays.back());
            checkCUDAError("kernBlockScan failed");

            for (int i = scanArrays.size() - 2; i >= 0; i--) {
                int arrayLen = scanArrayLens[i];
                kernAddSums << < arrayLen / maxBlockSize, maxBlockSize >> > (scanArrays[i], scanArrays[i + 1]);
                checkCUDAError("kernAddSums failed");
            }

            int* d_indices = scanArrays[0];


            kernScatter << <topArrayLen / maxBlockSize, maxBlockSize >> > (n, d_odata, d_idata, d_bools, d_indices);
            checkCUDAError("kernScatter failed");

            timer().endGpuTimer();

            hipDeviceSynchronize();

            int compactLen;
            hipMemcpy(&compactLen, &d_indices[topArrayLen - 1], sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy compactLen failed");
            int lastBool;
            hipMemcpy(&lastBool, &d_bools[topArrayLen - 1], sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy lastBool failed");
            compactLen += lastBool;

            hipMemcpy(odata, d_odata, compactLen * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy d_odata failed");

            for (int i = 0; i < scanArrays.size(); i++) {
                hipFree(scanArrays[i]);
                checkCUDAError("hipFree scanArrays[i] failed");
            }
            hipFree(d_bools);
            checkCUDAError("hipFree d_bools failed");
            hipFree(d_idata);
            checkCUDAError("hipFree d_idata failed");
            hipFree(d_odata);
            checkCUDAError("hipFree d_odata failed");

            return compactLen;
        }
    }
}
